#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
// #define CHECK(call){
//     const hipError_t error = call;
//     if(error != hipSuccess){
//         printf("Error: %s:%d",__FILE__,__LINE__);
//         printf("code:%d, reason:%s\n",error,hipGetErrorString(error));
//         exit(1);
//     }
// }

// block = N
__global__ void sumArraysOnDevice(float *A, float *B, float *C, const int N){
    int idx = threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

// grid = N
__global__ void sumArraysOnDeviceGrid(float *A,float *B,float *C, const int N){
    int idx = blockIdx.x;
    C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysOnDeviceCommon(float *A, float *B, float *C, const int N){
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    C[idx] = A[idx] + B[idx];
}

void sumArraysOnHost(float *A, float *B, float *C, const int N){
    for(int i =0; i < N; i++){
        C[i] = A[i] + B[i];
    }
}

void checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0e-8;
    bool match = true;
    for(int i =0; i< N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = false;
            printf("Result do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n",hostRef,gpuRef,i);
            break;
        }
    } 
    if(match) {
         printf("Result match!\n");
    }
}

void initialData(float *ip, int size){
    time_t t;
    srand((unsigned int) time(&t));

    for(int i =0; i < size; i++){
        ip[i] = (float)(rand() & 0xFF)/10.0f;
    }
}



int main(){

    int dev = 0;
    hipSetDevice(dev);
    
    int nElem = 32;
    printf("Vector size %d\n",nElem);
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *h_C,*h_Ref;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);
    h_Ref = (float *)malloc(nBytes);

    initialData(h_A,nElem);
    initialData(h_B,nElem);

    memset(h_C,0,nBytes);
    memset(h_Ref,0,nBytes);

    // cpu exec
    sumArraysOnHost(h_A,h_B,h_Ref,nElem);



    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A,nBytes);
    hipMalloc((float **)&d_B,nBytes);
    hipMalloc((float **)&d_C,nBytes);


    // cp data from cpu to gpu
    hipMemcpy(d_A,h_A,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,nBytes,hipMemcpyHostToDevice);
    
    int iLen = 128;
    dim3 block(iLen);
    dim3 grid((nElem + block.x -1)/block.x);

    // exec gpu 
    // sumArrasOnDevice(h_C,h_C,h_C,nElem);
    //sumArrasOnDevice<<<1,nElem>>>(d_A,d_B,d_C,nElem);
    // wait all device finish! hipMemcpy 内部还有同步，因此此处不需要加上 block
    // hipDeviceReset();
    sumArraysOnDeviceCommon<<<grid,block>>>(d_A,d_B,d_C,nElem);
    hipDeviceSynchronize();

    //
    hipMemcpy(h_C,d_C,nBytes,hipMemcpyDeviceToHost); 

    checkResult(h_Ref,h_C,nElem);

    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}