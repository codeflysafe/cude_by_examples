//
// Created by sjhuang on 2021/8/21.
//

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 100000

__global__ void vector_add(const float *a, const float *b, float *output,int n){
    for(int i =0; i < n; i++){
        output[i] = a[i] + b[i];
    }
}

void vector_output(float *output, int n){
    for(int i =0; i < n; i++){
        printf("output[%d] is %f",i,output[i]);
    }
}

int main(){
    // malloc memory
    float *a,*b,*output;
    a = (float *)malloc(sizeof (float )*N);
    b = (float *)malloc(sizeof (float )*N);
    output = (float*)malloc(sizeof(float) * N);
    // Initialize array
    for(int i =0; i < N; i++){
        a[i] = 1.0f, b[i] = 2.0f;
    }

    float *d_a,*d_b,*d_output;
    // Device Memory malloc
    hipMalloc(&d_a,sizeof (float )*N);
    hipMalloc(&d_b,sizeof (float )*N);
    hipMalloc(&d_output,sizeof (float )*N);
    // Transfer host data to device data
    hipMemcpy(d_a,a,sizeof (float )*N,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof (float )*N,hipMemcpyHostToDevice);

    // operate
    vector_add<<<1,1>>>(d_a,d_b,d_output,N);
    hipDeviceSynchronize();
    hipMemcpy(output,d_output,sizeof (float )*N,hipMemcpyDeviceToHost);

    // output
    printf("out[0] = %f\n", output[0]);
    printf("PASSED\n");
//
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_output);
    free(a);
    free(b);
    free(output);
    return 0;
}