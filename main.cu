
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
#define N 1000

#define MAX_ERR 1e-6

__global__ void add(int *a, int b, int c) {
   *a = b + c;
}


int main(){

    int count = 0;
    hipGetDevice(&count);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,count);
    int code = hipChooseDevice(&count,&prop);
    printf("%d,%d\n",code,count);
    int a = 0;
    int *dev_a;
    hipMalloc((void **)&dev_a,sizeof(int));

    add<<<1,1>>>(dev_a,3,5);
    hipDeviceSynchronize();
    hipMemcpy(&a,dev_a, sizeof(int),hipMemcpyDeviceToHost);

    printf("3 + 5  = %d\n",a);

    hipFree(dev_a);
    return 0;
}

